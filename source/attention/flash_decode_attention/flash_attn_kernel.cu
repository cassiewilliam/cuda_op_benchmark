#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::half_t, 64>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim64<cutlass::half_t>(params, stream);
}

template<>
void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128<cutlass::half_t>(params, stream);
}


template void run_mha_fwd_splitkv_dispatch<cutlass::half_t, 64>(Flash_fwd_params &params, hipStream_t stream);
template void run_mha_fwd_splitkv_dispatch<cutlass::half_t, 128>(Flash_fwd_params &params, hipStream_t stream);